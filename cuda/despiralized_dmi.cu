#include "hip/hip_runtime.h"
#include <stdint.h>
#include "exchange.h"
#include "float3.h"
#include "stencil.h"
#include "amul.h"

//Despiralized Exchange +  Despiralzied Dzyaloshinskii-Moriya interaction for bulk material.

// Energy:
// E = (A nabla m)² + D sin(qy) (MydzMz - MzdzMy +MydxMx -MxdxMz) + D cos(qy) (MzdxMy - MydxMz + MydxMx -Mxdzmy) + D²/4A My²
// q=D/2A
// Effective field:
//
// 	Hx = 2A/Bs nabla²Mx + 2D/Bs sin(qy) dyMy + 2D/Bs cos(qy) dzMy
// 	Hy = 2A/Bs nabla²My - 2D/Bs sin(qy) (dzMz + dxMx) - 2D/Bs cos(qy) (dzMx - dxMz) - D²/2ABs My
// 	Hz = 2A/Bs nabla²Mz + 2D/Bs sin(qy) dzMy - 2D/Bs cos(qy) dxMy
//
// Boundary conditions:
//
// 	        2A dxMx = 0
// 	 D Mz + 2A dxMy = 0
// 	-D My + 2A dxMz = 0
//
// 	-D Mz + 2A dyMx = 0
// 	        2A dyMy = 0
// 	 D Mx + 2A dyMz = 0
//
// 	 D My + 2A dzMx = 0
// 	-D Mx + 2A dzMy = 0
// 	        2A dzMz = 0
//
extern "C" __global__ void
adddmibulk(float* __restrict__ Hx, float* __restrict__ Hy, float* __restrict__ Hz,
           float* __restrict__ mx, float* __restrict__ my, float* __restrict__ mz,
           float* __restrict__ Ms_, float Ms_mul,
           float* __restrict__ aLUT2d, float* __restrict__ DLUT2d,
           uint8_t* __restrict__ regions,
           float cx, float cy, float cz, int Nx, int Ny, int Nz, uint8_t PBC, uint8_t OpenBC) {

    int ix = blockIdx.x * blockDim.x + threadIdx.x;
    int iy = blockIdx.y * blockDim.y + threadIdx.y;
    int iz = blockIdx.z * blockDim.z + threadIdx.z;

    if (ix >= Nx || iy >= Ny || iz >= Nz) {
        return;
    }

    int I = idx(ix, iy, iz);                      // central cell index
    float3 h = make_float3(0.0,0.0,0.0);          // add to H
    float3 m0 = make_float3(mx[I], my[I], mz[I]); // central m
    uint8_t r0 = regions[I];
    int i_;                                       // neighbor index

    if(is0(m0)) {
        return;
    }

    // x derivatives (along length)
    {
        float3 m1 = make_float3(0.0f, 0.0f, 0.0f);     // left neighbor
        i_ = idx(lclampx(ix-1), iy, iz);               // load neighbor m if inside grid, keep 0 otherwise
        if (ix-1 >= 0 || PBCx) {
            m1 = make_float3(mx[i_], my[i_], mz[i_]);
        }
        int r1 = is0(m1)? r0 : regions[i_];
        float A = aLUT2d[symidx(r0, r1)];
        float D = DLUT2d[symidx(r0, r1)];
        float D_2A = D/(2.0f*A);
        if (!is0(m1) || !OpenBC){                      // do nothing at an open boundary
            if (is0(m1)) {                             // neighbor missing
                m1.x = m0.x;
                m1.y = m0.y - (-cx * D_2A * m0.z);
                m1.z = m0.z + (-cx * D_2A * m0.y);
            }
            h   += (2.0f*A/(cx*cx)) * (m1 - m0);       // exchange
            h.x += (D/cx)*sin(D_2A*cy*iy)*(-m1.y);
            h.y -= (D/cx)*(sin(D_2A*cy*iy)*(-m1.x) - cos(D_2A*cy*iy)*(-m1.z));
            h.z -= (D/cx)*cos(D_2A*cy*iy)*(-m1.y);
        }
    }


    {
        float3 m2 = make_float3(0.0f, 0.0f, 0.0f);     // right neighbor
        i_ = idx(hclampx(ix+1), iy, iz);
        if (ix+1 < Nx || PBCx) {
            m2 = make_float3(mx[i_], my[i_], mz[i_]);
        }
        int r1 = is0(m2)? r0 : regions[i_];
        float A = aLUT2d[symidx(r0, r1)];
        float D = DLUT2d[symidx(r0, r1)];
        float D_2A = D/(2.0f*A);
        if (!is0(m2) || !OpenBC){
            if (is0(m2)) {
                m2.x = m0.x;
                m2.y = m0.y - (+cx * D_2A * m0.z);
                m2.z = m0.z + (+cx * D_2A * m0.y);
            }
            h   += (2.0f*A/(cx*cx)) * (m2 - m0);
            h.x += (D/cx)*sin(D_2A*cy*iy)*(m2.y);
            h.y -= (D/cx)*(sin(D_2A*cy*iy)*(m2.x) - cos(D_2A*cy*iy)*(m2.z));
            h.z -= (D/cx)*cos(D_2A*cy*iy)*(m2.y);
        }
    }

    
    // z derivatives
    // only take vertical derivative for 3D sim
    if (Nz != 1) {
        // bottom neighbor
        {
            float3 m1 = make_float3(0.0f, 0.0f, 0.0f);
            i_ = idx(ix, iy, lclampz(iz-1));
            if (iz-1 >= 0 || PBCz) {
                m1 = make_float3(mx[i_], my[i_], mz[i_]);
            }
            int r1 = is0(m1)? r0 : regions[i_];
            float A = aLUT2d[symidx(r0, r1)];
            float D = DLUT2d[symidx(r0, r1)];
            float D_2A = D/(2.0f*A);
            if (!is0(m1) || !OpenBC){
                if (is0(m1)) {
                    m1.x = m0.x - (-cz * D_2A * m0.y);
                    m1.y = m0.y + (-cz * D_2A * m0.x);
                    m1.z = m0.z;
                }
                h   += (2.0f*A/(cz*cz)) * (m1 - m0);
                h.x += (D/cz)*cos(D_2A*cy*iy)*(-m1.y);
                h.y -= (D/cz)*(sin(D_2A*cy*iy)*(-m1.z) + cos(D_2A*cy*iy)*(-m1.z));
                h.z += (D/cz)*sin(D_2A*cy*iy)*(-m1.y);
            }
        }

        // top neighbor
        {
            float3 m2 = make_float3(0.0f, 0.0f, 0.0f);
            i_ = idx(ix, iy, hclampz(iz+1));
            if (iz+1 < Nz || PBCz) {
                m2 = make_float3(mx[i_], my[i_], mz[i_]);
            }
            int r1 = is0(m2)? r0 : regions[i_];
            float A = aLUT2d[symidx(r0, r1)];
            float D = DLUT2d[symidx(r0, r1)];
            float D_2A = D/(2.0f*A);
            if (!is0(m2) || !OpenBC){
                if (is0(m2)) {
                    m2.x = m0.x - (+cz * D_2A * m0.y);
                    m2.y = m0.y + (+cz * D_2A * m0.x);
                    m2.z = m0.z;
                }
                h   += (2.0f*A/(cz*cz)) * (m2 - m0);
                h.x += (D/cz)*cos(D_2A*cy*iy)*(m2.y);
                h.y -= (D/cz)*(sin(D_2A*cy*iy)*(m2.z) + cos(D_2A*cy*iy)*(m2.z));
                h.z += (D/cz)*sin(D_2A*cy*iy)*(m2.y);
            }
        }
    }

    // write back, result is H + Hdmi + Hex
    float invMs = inv_Msat(Ms_, Ms_mul, I);
    Hx[I] += h.x*invMs;
    Hy[I] += h.y*invMs;
    Hz[I] += h.z*invMs;
}

// Note on boundary conditions.
//
// We need the derivative and laplacian of m in point A, but e.g. C lies out of the boundaries.
// We use the boundary condition in B (derivative of the magnetization) to extrapolate m to point C:
// 	m_C = m_A + (dm/dx)|_B * cellsize
//
// When point C is inside the boundary, we just use its actual value.
//
// Then we can take the central derivative in A:
// 	(dm/dx)|_A = (m_C - m_D) / (2*cellsize)
// And the laplacian:
// 	lapl(m)|_A = (m_C + m_D - 2*m_A) / (cellsize^2)
//
// All these operations should be second order as they involve only central derivatives.
//
//    ------------------------------------------------------------------ *
//   |                                                   |             C |
//   |                                                   |          **   |
//   |                                                   |        ***    |
//   |                                                   |     ***       |
//   |                                                   |   ***         |
//   |                                                   | ***           |
//   |                                                   B               |
//   |                                               *** |               |
//   |                                            ***    |               |
//   |                                         ****      |               |
//   |                                     ****          |               |
//   |                                  ****             |               |
//   |                              ** A                 |               |
//   |                         *****                     |               |
//   |                   ******                          |               |
//   |          *********                                |               |
//   |D ********                                         |               |
//   |                                                   |               |
//   +----------------+----------------+-----------------+---------------+
//  -1              -0.5               0               0.5               1
//                                 x
